
#include <hip/hip_runtime.h>
#ifdef OCTOTIGER_HAVE_CUDA

#include <hpx/executors/async.hpp>
#include <hpx/include/apply.hpp>
#include <buffer_manager.hpp>
#include <cuda_buffer_util.hpp>
#include "octotiger/options.hpp"
#include "octotiger/cuda_util/cuda_helper.hpp"
#include <cuda_runtime.h>
#include <stream_manager.hpp>

#include "octotiger/unitiger/hydro_impl/flux_kernel_interface.hpp"

#include <hpx/synchronization/once.hpp>

__device__ inline int flip_dim(const int d, const int flip_dim) {
		int dims[3];
		int k = d;
		for (int dim = 0; dim < 3; dim++) {
			dims[dim] = k % 3;
			k /= 3;
		}
		k = 0;
		dims[flip_dim] = 2 - dims[flip_dim];
		for (int dim = 0; dim < 3; dim++) {
			k *= 3;
			k += dims[2 - dim];
		}
		return k;
}

__device__ const int faces[3][9] = { { 12, 0, 3, 6, 9, 15, 18, 21, 24 }, { 10, 0, 1, 2, 9, 11,
			18, 19, 20 }, { 4, 0, 1, 2, 3, 5, 6, 7, 8 } };

__device__ const int xloc[27][3] = {
	/**/{ -1, -1, -1 }, { +0, -1, -1 }, { +1, -1, -1 },
	/**/{ -1, +0, -1 }, { +0, +0, -1 }, { 1, +0, -1 },
	/**/{ -1, +1, -1 }, { +0, +1, -1 }, { +1, +1, -1 },
	/**/{ -1, -1, +0 }, { +0, -1, +0 }, { +1, -1, +0 },
	/**/{ -1, +0, +0 }, { +0, +0, +0 }, { +1, +0, +0 },
	/**/{ -1, +1, +0 }, { +0, +1, +0 }, { +1, +1, +0 },
	/**/{ -1, -1, +1 }, { +0, -1, +1 }, { +1, -1, +1 },
	/**/{ -1, +0, +1 }, { +0, +0, +1 }, { +1, +0, +1 },
	/**/{ -1, +1, +1 }, { +0, +1, +1 }, { +1, +1, +1 } };

__device__ const double quad_weights[9] = { 16. / 36., 1. / 36., 4. / 36., 1. / 36., 4. / 36., 4.
			/ 36., 1. / 36., 4. / 36., 1. / 36. };

hpx::lcos::local::once_flag flag1;

__host__ void init_gpu_masks(bool *masks) {
  auto masks_boost = create_masks();
  cudaMemcpy(masks, masks_boost.data(), NDIM * 1000 * sizeof(bool), cudaMemcpyHostToDevice);
}

__host__ const bool* get_gpu_masks(void) {
    static bool *masks = recycler::recycle_allocator_cuda_device<bool>{}.allocate(NDIM * 1000);
    hpx::lcos::local::call_once(flag1, init_gpu_masks, masks);
    return masks;
}

__device__ const int offset = 0;
__device__ const int compressedH_DN[3] = {100, 10, 1};
__device__ const int face_offset = 27 * 1000;
__device__ const int dim_offset = 1000;

__global__ void
__launch_bounds__(900, 1)
 flux_cuda_kernel(const double * __restrict__ q_combined, const double * __restrict__ x_combined, double * __restrict__ f_combined,
    double * amax, int * amax_indices, int * amax_d, const bool * __restrict__ masks, const double omega, const double dx, const double A_, const double B_, const double fgamma, const double de_switch_1) {
  __shared__ double sm_amax[900];
  __shared__ int sm_d[900];
  __shared__ int sm_i[900];

  // 3 dim 1000 i workitems
  const int dim = blockIdx.z;
  const int index = threadIdx.x * 100 + threadIdx.y * 10 + threadIdx.z + 100;
  int tid = index - 100;   
  double mask = masks[index + dim * dim_offset];
  //if(tid == 0)
  // printf("starting...");
  const int nf = 15;

  double local_f[15] = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
  double local_x[3] = {0.0, 0.0, 0.0};
  double local_vg[3] = {0.0, 0.0, 0.0};
  for (int f = 0; f < nf; f++) {
      f_combined[dim * 15 * 1000 + f * 1000 + index] = 0.0;
  }

  double current_amax = 0.0;
  int current_d = 0;
  if(mask != 0.0) {
    for (int fi = 0; fi < 9; fi++) {    // 9
      double this_ap = 0.0, this_am = 0.0;    // tmps
      const int d = faces[dim][fi];
      const int flipped_dim = flip_dim(d, dim);
      for (int dim = 0; dim < 3; dim++) {
          local_x[dim] = x_combined[dim * 1000 + index] + (0.5 * xloc[d][dim] * dx);
      }
      local_vg[0] = -omega * (x_combined[1000 + index] + 0.5 * xloc[d][1] * dx);
      local_vg[1] = +omega * (x_combined[index] + 0.5 * xloc[d][0] * dx);
      local_vg[2] = 0.0;
      inner_flux_loop2<double>(omega, nf, A_, B_, q_combined, local_f, local_x, local_vg,
        this_ap, this_am, dim, d, dx, fgamma, de_switch_1, dim_offset * d + index, dim_offset * flipped_dim - compressedH_DN[dim] + index, face_offset);
      this_ap *= mask;
      this_am *= mask;
      const double amax_tmp = max_wrapper(this_ap, (-this_am));
      if (amax_tmp > current_amax) {
        current_amax = amax_tmp;
        current_d = d;
      }
      for (int f = 0; f < nf; f++) {
        f_combined[dim * 15 * 1000 + f * 1000 + index] += quad_weights[fi] * local_f[f];
      }
   }
 }

 // Find maximum:
 sm_amax[tid] = current_amax;
 sm_d[tid] = current_d;
 sm_i[tid] = tid;
 __syncthreads();
 // First step as we do not have multiples of 2
 if(tid < 450) {
   if (sm_amax[tid + 450 ] > sm_amax[tid]) {
     sm_amax[tid] = sm_amax[tid + 450];
     sm_d[tid] = sm_d[tid + 450];
     sm_i[tid] = sm_i[tid + 450];
   }
 }
 __syncthreads();
 // Max reduction with multiple warps
 for (int tid_border = 256; tid_border >= 32; tid_border /= 2) {
   if(tid < tid_border) {
     if (sm_amax[tid + tid_border] > sm_amax[tid]) {
       sm_amax[tid] = sm_amax[tid + tid_border];
       sm_d[tid] = sm_d[tid + tid_border];
       sm_i[tid] = sm_i[tid + tid_border];
     }
   }
   __syncthreads();
 }
 // Max reduction within one warps
 for (int tid_border = 16; tid_border >= 1; tid_border /= 2) {
   if(tid < tid_border) {
     if (sm_amax[tid + tid_border] > sm_amax[tid]) {
       sm_amax[tid] = sm_amax[tid + tid_border];
       sm_d[tid] = sm_d[tid + tid_border];
       sm_i[tid] = sm_i[tid + tid_border];
     }
   }
 }

 if (tid == 0) {
   amax[dim] = sm_amax[0];
   amax_indices[dim] = sm_i[0];
   amax_d[dim] = sm_d[0];
 }
 return;
}

timestep_t launch_flux_cuda(const std::vector<double, recycler::recycle_allocator_cuda_host<double>> &combined_q, std::vector<double, recycler::recycle_allocator_cuda_host<double>> &combined_f, hydro::x_type& X,
    safe_real omega, const size_t nf_) {
    timestep_t ts;

    // Check availability
    // TODO(daissgr) Why is this segfaulting at the end of the simulation despite not being called?
  /*  bool avail = stream_pool::interface_available<hpx::cuda::experimental::cuda_executor,
                 pool_strategy>(opts().cuda_buffer_capacity);
  
    if (!avail) {
      std::cerr << "Warning, high GPU load in flux detected... This shouldn't happen" << std::endl;
    } 

    size_t device_id =
      stream_pool::get_next_device_id<hpx::cuda::experimental::cuda_executor,
      pool_strategy>();*/
    size_t device_id = 0;

    stream_interface<hpx::cuda::experimental::cuda_executor, pool_strategy> executor;

    recycler::cuda_device_buffer<double> device_q(15 * 27 * 10 * 10 * 10 + 32, device_id);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
    cudaMemcpyAsync, device_q.device_side_buffer,
    combined_q.data(), (15 * 27 * 10 * 10 * 10 + 32) * sizeof(double), cudaMemcpyHostToDevice);

    std::vector<double, recycler::recycle_allocator_cuda_host<double>> combined_x(NDIM * 1000 + 32);
    convert_x_structure(X, combined_x);
    const cell_geometry<3, 8> geo;
    double dx = X[0][geo.H_DNX] - X[0][0];
    recycler::cuda_device_buffer<double> device_x(NDIM * 1000 + 32, device_id);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
    cudaMemcpyAsync, device_x.device_side_buffer,
    combined_x.data(), (NDIM * 1000 + 32) * sizeof(double), cudaMemcpyHostToDevice);

    recycler::cuda_device_buffer<double> device_f(NDIM * 15 * 1000 + 32, device_id);
    const bool *masks = get_gpu_masks();

    recycler::cuda_device_buffer<double> device_amax(NDIM);
    recycler::cuda_device_buffer<int> device_amax_indices(NDIM);
    recycler::cuda_device_buffer<int> device_amax_d(NDIM);
    double A_ = physics<NDIM>::A_;
    double B_ = physics<NDIM>::B_;
    double fgamma = physics<NDIM>::fgamma_;
    double de_switch_1 = physics<NDIM>::de_switch_1;

    dim3 const grid_spec(1, 1, 3);
    dim3 const threads_per_block(9, 10, 10);
    void* args[] = {&(device_q.device_side_buffer),
      &(device_x.device_side_buffer), &(device_f.device_side_buffer), &(device_amax.device_side_buffer),
      &(device_amax_indices.device_side_buffer), &(device_amax_d.device_side_buffer), &masks, &omega, &dx, &A_, &B_, &fgamma, &de_switch_1};
    executor.post(
    cudaLaunchKernel<decltype(flux_cuda_kernel)>,
    flux_cuda_kernel, grid_spec, threads_per_block, args, 0);

    // Move data to host
    std::vector<double, recycler::recycle_allocator_cuda_host<double>> amax(NDIM);
    std::vector<int, recycler::recycle_allocator_cuda_host<int>> amax_indices(NDIM);
    std::vector<int, recycler::recycle_allocator_cuda_host<int>> amax_d(NDIM);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, amax.data(),
               device_amax.device_side_buffer, NDIM * sizeof(double),
               cudaMemcpyDeviceToHost);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, amax_indices.data(),
               device_amax_indices.device_side_buffer, NDIM * sizeof(int),
               cudaMemcpyDeviceToHost);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, amax_d.data(),
               device_amax_d.device_side_buffer, NDIM * sizeof(int),
               cudaMemcpyDeviceToHost);
    auto fut = hpx::async(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, combined_f.data(), device_f.device_side_buffer,
               (NDIM * 15 * 1000 + 32) * sizeof(double), cudaMemcpyDeviceToHost);
    fut.get();

    // Find Maximum
    size_t current_dim = 0;
    for (size_t dim_i = 1; dim_i < NDIM; dim_i++) {
      if (amax[dim_i] > amax[current_dim]) { 
        current_dim = dim_i;
      }
    }
    //std::cin.get();
    std::vector<double> URs(nf_), ULs(nf_);
    const size_t current_max_index = amax_indices[current_dim];
    const size_t current_d = amax_d[current_dim];
    ts.a = amax[current_dim];
    ts.x = combined_x[current_max_index];
    ts.y = combined_x[current_max_index + 1000];
    ts.z = combined_x[current_max_index + 2000];
    const auto flipped_dim = geo.flip_dim(current_d, current_dim);
    constexpr int compressedH_DN[3] = {100, 10, 1};
    for (int f = 0; f < nf_; f++) {
        URs[f] = combined_q[current_max_index + f * face_offset + dim_offset * current_d];
        ULs[f] = combined_q[current_max_index - compressedH_DN[current_dim] + f * face_offset +
            dim_offset * flipped_dim];
    }
    ts.ul = std::move(ULs);
    ts.ur = std::move(URs);
    ts.dim = current_dim;
    return ts;
}


#endif
